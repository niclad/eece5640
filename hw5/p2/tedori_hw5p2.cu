// mpi authors


#include <hip/hip_runtime.h>
#include <algorithm> // swap
#include <cstdio>
#include <fstream> // file io
#include <iomanip>
#include <iostream> // io
#include <cmath>
#include <sstream> // string stream
#include <string>  // strings
#include <time.h>

using namespace std;

const int VERT = 317080;   // from http://snap.stanford.edu/data/com-DBLP.html
const int EDGES = 1049866; // number of edges. from above
const int TOP = 10;        // top number of authors to determine

void readFileCSR(string fileName, int *rowIdx);
__global__ void countAuthors(int *d_rowIdx, int *d_counts, int n);
__global__ void topAuthors(int *d_counts, int *d_max, int top, int n);
__global__ void coauthorVolume(int *d_dist, int *d_counts, int max, int n);
void determineTop(int *authors, int *topAuth, int *topCounts);
void sort(int *id, int *val, int len);
void generateData(int *dist, int distSize);
double CLOCK();

int main(int argc, char *argv[])
{
    string fileName = "dblp-co-authors.txt";             // file name with data
    int *h_rowIdx, *h_counts, *h_max, *h_tCnts, *h_dist; // host authors
    int *d_rowIdx, *d_counts, *d_max, *d_dist;           // device authors
    size_t rowBytes, cntBytes, maxBytes;                 // size (in bytes) of each array
    double start, finish, total = 0;                     // timing variables
    int max;                                             // maximum number of co-authors

    // determine size of device arrays
    rowBytes = (VERT + 1) * sizeof(int);
    cntBytes = VERT * sizeof(int);
    maxBytes = TOP * sizeof(int);

    // allocate memory on host
    h_rowIdx = new int[VERT + 1]; // host row counts (CSR format)
    h_counts = new int[VERT];
    h_max = new int[TOP];
    h_tCnts = new int[TOP];
    hipMalloc(&d_rowIdx, rowBytes); // allocate memory device
    hipMalloc(&d_counts, cntBytes);
    hipMalloc(&d_max, maxBytes);

    // initialize row offset array
    for (int i = 0; i < VERT + 1; i++)
    {
        h_rowIdx[i] = 0;
    }

    // get device properties
    hipDeviceProp_t props;               // devices properties
    hipGetDeviceProperties(&props, 0); // get the device properties
    cout << "GPU: " << props.name << ": " << props.major << "." << props.minor << endl;

    start = CLOCK();
    readFileCSR(fileName, h_rowIdx);
    finish = CLOCK() - start;
    total += finish;
    cout << "File read time: " << finish / 1000 << " sec" << endl;

    for (int i = 0; i < VERT; i++) // initalize host arrays
        h_counts[i] = 0;
    //for (int i = 0; i < TOP; i++)
        //h_max[i] = i;

    hipMemcpy(d_rowIdx, h_rowIdx, rowBytes, hipMemcpyHostToDevice); // copy array to device
    hipMemcpy(d_counts, h_counts, cntBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_max, h_max, maxBytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;
    blockSize = 1024;                                    // number of threads in each thread block
    gridSize = (int)ceil((float)(VERT + 1) / blockSize); // number of thread blocks in a grid

    // COUNT AUTHORS' CO-AUTHORS
    start = CLOCK();
    // execute the kernel
    countAuthors<<<gridSize, blockSize>>>(d_rowIdx, d_counts, VERT);
    finish = CLOCK() - start;
    total += finish;
    cout << "GPU author count time: " << finish / 1000 << " sec" << endl;

    hipMemcpy(h_counts, d_counts, cntBytes, hipMemcpyDeviceToHost); // copy data back to host

    // DETERMINE TOP X AUTHORS WITH MOST CO-AUTHORS
    start = CLOCK();
    determineTop(h_counts, h_max, h_tCnts);
    sort(h_max, h_tCnts, TOP);
    finish = CLOCK() - start;
    total += finish;
    cout << "Max co-auth. search time: " << finish / 1000 << " sec" << endl;

    // GENERATE GRAPHING DATA
    max = h_tCnts[0];                       // get the maximum no. of co-authors
    h_dist = new int[max];                  // allocate memory on host
    hipMalloc(&d_dist, max * sizeof(int)); // allocate memory on device

    // initialize array
    for (int i = 0; i < max; i++)
    {
        h_dist[i] = 0;
    }

    hipMemcpy(d_dist, h_dist, max * sizeof(int), hipMemcpyHostToDevice); // copy array to device

    start = CLOCK();
    gridSize = (int)ceil((float)VERT / blockSize);                        // update gridSize
    coauthorVolume<<<gridSize, blockSize>>>(d_dist, d_counts, max, VERT); // execute the kernel
    finish = CLOCK() - start;
    total += finish;
    cout << "GPU data generation time: " << finish / 1000 << " sec" << endl;

    hipMemcpy(h_dist, d_dist, max * sizeof(int), hipMemcpyDeviceToHost); // copy array to host

    // DISPLAY RUNNING INFO
    cout << "***************************" << endl; 
    cout << "FINAL RESULTS:" << endl;
     for (int i = 0; i < TOP; i++)
    {
        cout << "auth" << setw(6) << h_max[i] + 1 << ", count " << h_tCnts[i] << endl;
    }
    cout << "First 10 distribution values:" << endl;
    for (int i = 0; i < TOP; i++)
    {
        cout << h_dist[i] << endl;
    }
    cout << "Last: " << h_dist[342] << endl; // display the last value

    generateData(h_dist, max); // generate the graphing data

    cout << "Cumulative running time: " << total / 1000 << " sec" << endl;

    hipFree(d_rowIdx); // free device memory
    hipFree(d_counts);
    hipFree(d_max);
    hipFree(d_dist);
    delete[] h_rowIdx; // free host memory
    delete[] h_counts;
    delete[] h_max;
    delete[] h_dist;
}

/**
 * @brief Read a file, generating a CSR sparse matrix
 * 
 * @param fileName      File name to read from
 * @param rowIdx        Row index (offset)
 * @param colIdx        Column index
 * @param counts        Coauthor counts
 */
void readFileCSR(string fileName, int *rowIdx)
{
    ifstream fin(fileName.c_str()); // open the input file
    size_t found;                   // check for the first character
    string line, tempAuth;
    char delim = ' '; // delimiter for the data
    int author;       // author value
    int adjIdx[2];    // adjaceny matrix value indices

    int count = 0;
    while (fin)
    {
        // check for line being a comment
        getline(fin, line);
        found = line.find_first_not_of(" \t");

        // check for comments in the file. comments appear to start with '%'
        if (found != string::npos)
        {
            // if the line is a comment, move onto the next one.
            if (line[found] == '%')
                continue;
        }

        stringstream s(line);
        int i = 0;
        while (getline(s, tempAuth, delim))
        {
            author = stoi(tempAuth);
            adjIdx[i] = author;
            i++;
            //cout << author << "\t";
        }
        //cout << adjIdx[0] << " " << adjIdx[1] << endl;  // DEBUGGING
        int idx0 = adjIdx[0];
        int idx1 = adjIdx[1];

        rowIdx[idx0]++;
        rowIdx[idx1]++; // = rowIdx[idx1-1] + rowIdx[idx1] + 1;

        if (fin.eof())
            break;

        count++;
    }
    fin.close();

    // cumulatively sum the elements
    for (int i = 2; i < VERT + 2; i++)
    {
        rowIdx[i] = rowIdx[i] + rowIdx[i - 1];
    }
    cout << "Lines processed: " << count << endl;
}

/**
 * @brief CUDA kernel to get the top authors and retrieve author counts
 * 
 * @param d_rowIdx 
 * @param d_counts 
 * @param d_max 
 * @param n 
 */
__global__ void countAuthors(int *d_rowIdx, int *d_counts, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // get global thread id
    int countVal;                                   // total co-authors

    if (id < n) // Make sure the threads don't index out of range
    {
        countVal = d_rowIdx[id + 1] - d_rowIdx[id]; // calculate the number of co-authors
        d_counts[id] = countVal;
    }
}

__global__ void topAuthors(int *d_counts, int *d_max, int top, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // get global thread id
    int stride = blockDim.x * gridDim.x;            // stride length
    int smallest = top - 1;                         // index of smallest member of d_max
    int curr = 0;
    __shared__ int t_top[20];
    __shared__ int t_cnt[20];

    for (int i = 0; i < top; i++)
    {
        t_top[i] = i;
        t_cnt[i] = 0;
    }

    if (id < n)
    {
        for (int i = id; i < n; i += stride)
        {
            curr = d_counts[i];
            if (curr > d_counts[t_top[smallest]])
            {
                t_top[smallest] = i;
                t_cnt[smallest] = d_counts[i];
                for (int j = threadIdx.x; j < 20; j += 2) // find new smallest
                {
                    int comp1 = t_top[j];
                    int comp2 = t_top[smallest];
                    if (d_counts[comp1] < d_counts[comp2])
                    {
                        smallest = j;
                    }
                }
            }
        }
    }

    if (threadIdx.x == 0)
    {
        printf("Thread: %d\n", threadIdx.x);
        for (int i = 0; i < 20; i++)
            printf("auth %d, count %d\n", (t_top[i] + 1), t_cnt[i]);
    }
}

/**
 * @brief 
 * 
 * @param d_dist 
 * @param d_counts 
 * @param max 
 */
__global__ void coauthorVolume(int *d_dist, int *d_counts, int max, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // get global thread id

    if (id < n)
    {
        int memLoc = d_counts[id] - 1;   // get the index that needs to be incremented
        atomicAdd((d_dist + memLoc), 1); // add 1 to the value at d_dist[memLoc]
    }
}

/**
 * @brief Detertime the top number of authors
 * 
 * @param authors       list of authors in order of id (index)
 * @param authPerProc   size of authors
 * @param topAuth       list of top authors
 * @param topCounts     counts associated with author at index
 * @param top           size of top list
 */
void determineTop(int *authors, int *topAuth, int *topCounts)
{
    int currCount;
    for (int i = 0; i < VERT; i++)
    {
        currCount = authors[i];

        for (int j = 0; j < TOP; j++)
        {
            if (currCount > topCounts[j])
            {
                // replace smallest author at the end.
                topAuth[TOP - 1] = i;
                topCounts[TOP - 1] = currCount;

                sort(topAuth, topCounts, TOP);
                break;
            }
        }
    }
}

/**
 * @brief Bubble sort
 * 
 * @param id    ids assoc with vals
 * @param val   values to sort
 * @param len   list length
 */
void sort(int *id, int *val, int len)
{
    bool swapped;

    for (int i = 0; i < len - 1; i++)
    {
        swapped = false;
        for (int j = 0; j < len - i - 1; j++)
        {
            if (val[j] < val[j + 1])
            {
                swap(val[j], val[j + 1]);
                swap(id[j], id[j + 1]);
                swapped = true;
            }
        }

        if (!swapped)
            break;
    }
}

/**
 * @brief Generate a data file from passed values
 * 
 * @param dist      Distribution
 * @param distSize  Size of the distribution
 */
void generateData(int *dist, int distSize)
{
    ofstream data("author_data-cuda.txt");

    for (int i = 0; i < distSize-1; i++)
    {
        data << i+1 << " " << dist[i] << "\n";
    }
    data << distSize << " " << dist[distSize-1];

    cout << "Data saved as \"author_data-cuda.txt\"" << endl;
}

/**
 * @brief Get a time point
 * 
 * @return double   The time point generated
 */
double CLOCK()
{
    struct timespec t;
    clock_gettime(CLOCK_MONOTONIC, &t);
    return (t.tv_sec * 1000) + (t.tv_nsec * 1e-6);
}