// mpi authors


#include <hip/hip_runtime.h>
#include <algorithm> // swap
#include <cstdio>
#include <fstream> // file io
#include <iomanip>
#include <iostream> // io
#include <cmath>
#include <sstream> // string stream
#include <string>  // strings
#include <time.h>

using namespace std;

const int VERT = 317080;   // from http://snap.stanford.edu/data/com-DBLP.html
const int EDGES = 1049866; // number of edges. from above
const int TOP = 10;        // top number of authors to determine

void readFileCSR(string fileName, int *rowIdx);
__global__ void countAuthors(int *d_rowIdx, int *d_counts, int n);
__global__ void topAuthors(int *d_counts, int *d_max, int top, int n);
void determineTop(int *authors, int *topAuth, int *topCounts);
void sort(int *id, int *val, int len);
double CLOCK();

int main(int argc, char *argv[])
{
    string fileName = "dblp-co-authors.txt";    // file name with data
    int *h_rowIdx, *h_counts, *h_max, *h_tCnts; // host authors
    int *d_rowIdx, *d_counts, *d_max;           // device authors
    size_t rowBytes, cntBytes, maxBytes;        // size (in bytes) of each array
    double start, finish, total = 0;            // timing variables

    // determine size of device arrays
    rowBytes = (VERT + 1) * sizeof(int);
    cntBytes = VERT * sizeof(int);
    maxBytes = TOP * sizeof(int);

    // allocate memory on host
    h_rowIdx = new int[VERT + 1]; // host row counts (CSR format)
    h_counts = new int[VERT];
    h_max = new int[TOP];
    h_tCnts = new int[TOP];
    hipMalloc(&d_rowIdx, rowBytes); // allocate memory device
    hipMalloc(&d_counts, cntBytes);
    hipMalloc(&d_max, maxBytes);

    // initialize row offset array
    for (int i = 0; i < VERT + 1; i++)
    {
        h_rowIdx[i] = 0;
    }

    // get device properties
    hipDeviceProp_t props;               // devices properties
    hipGetDeviceProperties(&props, 0); // get the device properties
    cout << "GPU: " << props.name << ": " << props.major << "." << props.minor << endl;

    start = CLOCK();
    readFileCSR(fileName, h_rowIdx);
    finish = CLOCK() - start;
    total += finish;
    cout << "File read time: " << finish / 1000 << " sec" << endl;

    for (int i = 0; i < VERT; i++) // initalize host arrays
        h_counts[i] = 0;
    //for (int i = 0; i < TOP; i++)
        //h_max[i] = i;

    hipMemcpy(d_rowIdx, h_rowIdx, rowBytes, hipMemcpyHostToDevice); // copy array to device
    hipMemcpy(d_counts, h_counts, cntBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_max, h_max, maxBytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;
    blockSize = 1024;                                    // number of threads in each thread block
    gridSize = (int)ceil((float)(VERT + 1) / blockSize); // number of thread blocks in a grid

    start = CLOCK();
    // execute the kernel
    countAuthors<<<gridSize, blockSize>>>(d_rowIdx, d_counts, VERT);
    finish = CLOCK() - start;
    total += finish;
    cout << "GPU author count time: " << finish / 1000 << " sec" << endl;

    //topAuthors<<<1, 2>>>(d_counts, d_max, TOP, VERT);
    hipMemcpy(h_rowIdx, d_rowIdx, rowBytes, hipMemcpyDeviceToHost); // copy data back to host
    hipMemcpy(h_counts, d_counts, cntBytes, hipMemcpyDeviceToHost);
    //cudaMemcpy(h_max, d_max, maxBytes, cudaMemcpyDeviceToHost);

    start = CLOCK();
    determineTop(h_counts, h_max, h_tCnts);
    sort(h_max, h_tCnts, TOP);
    finish = CLOCK() - start;
    total += finish;
    cout << "Max co-auth. search time: " << finish / 1000 << " sec" << endl;

    for (int i = 0; i < TOP; i++)
    {
        cout << "auth" << setw(6) << h_max[i] + 1 << ", count " << h_tCnts[i] << endl;
    }

    cout << "Total running time: " << total / 1000 << " sec" << endl;

    // display data on host arrays
    /*
    cout << "First 5 counts:" << endl;
    for (int i = 0; i < 5; i++)
        cout << h_counts[i] << endl;
*/
    //cout << "Max authors in no order:" << endl;
    //for (int i = 0; i < 1; i++)
    //    cout << h_max[i] << " " << h_counts[h_max[i]] << endl;

    hipFree(d_rowIdx); // free device memory
    hipFree(d_counts);
    hipFree(d_max);
    delete[] h_rowIdx; // free host memory
    delete[] h_counts;
    delete[] h_max;
}

/**
 * @brief Read a file, generating a CSR sparse matrix
 * 
 * @param fileName      File name to read from
 * @param rowIdx        Row index (offset)
 * @param colIdx        Column index
 * @param counts        Coauthor counts
 */
void readFileCSR(string fileName, int *rowIdx)
{
    ifstream fin(fileName.c_str()); // open the input file
    size_t found;                   // check for the first character
    string line, tempAuth;
    char delim = ' '; // delimiter for the data
    int author;       // author value
    int adjIdx[2];    // adjaceny matrix value indices

    int count = 0;
    while (fin)
    {
        // check for line being a comment
        getline(fin, line);
        found = line.find_first_not_of(" \t");

        // check for comments in the file. comments appear to start with '%'
        if (found != string::npos)
        {
            // if the line is a comment, move onto the next one.
            if (line[found] == '%')
                continue;
        }

        stringstream s(line);
        int i = 0;
        while (getline(s, tempAuth, delim))
        {
            author = stoi(tempAuth);
            adjIdx[i] = author;
            i++;
            //cout << author << "\t";
        }
        //cout << adjIdx[0] << " " << adjIdx[1] << endl;  // DEBUGGING
        int idx0 = adjIdx[0];
        int idx1 = adjIdx[1];

        rowIdx[idx0]++;
        rowIdx[idx1]++; // = rowIdx[idx1-1] + rowIdx[idx1] + 1;

        if (fin.eof())
            break;

        count++;
    }
    fin.close();

    // cumulatively sum the elements
    for (int i = 2; i < VERT + 2; i++)
    {
        rowIdx[i] = rowIdx[i] + rowIdx[i - 1];
    }
    cout << "Lines processed: " << count << endl;
}

/**
 * @brief CUDA kernel to get the top authors and retrieve author counts
 * 
 * @param d_rowIdx 
 * @param d_counts 
 * @param d_max 
 * @param n 
 */
__global__ void countAuthors(int *d_rowIdx, int *d_counts, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // get global thread id
    int countVal;                                   // total co-authors

    if (id < n)
    {
        countVal = d_rowIdx[id + 1] - d_rowIdx[id];
        d_counts[id] = countVal;
    }
}

__global__ void topAuthors(int *d_counts, int *d_max, int top, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // get global thread id
    int stride = blockDim.x * gridDim.x;            // stride length
    int smallest = top - 1;                         // index of smallest member of d_max
    int curr = 0;
    __shared__ int t_top[20];
    __shared__ int t_cnt[20];

    for (int i = 0; i < top; i++)
    {
        t_top[i] = i;
        t_cnt[i] = 0;
    }

    if (id < n)
    {
        for (int i = id; i < n; i += stride)
        {
            curr = d_counts[i];
            if (curr > d_counts[t_top[smallest]])
            {
                t_top[smallest] = i;
                t_cnt[smallest] = d_counts[i];
                for (int j = threadIdx.x; j < 20; j += 2) // find new smallest
                {
                    int comp1 = t_top[j];
                    int comp2 = t_top[smallest];
                    if (d_counts[comp1] < d_counts[comp2])
                    {
                        smallest = j;
                    }
                }
            }
        }
    }

    if (threadIdx.x == 0)
    {
        printf("Thread: %d\n", threadIdx.x);
        for (int i = 0; i < 20; i++)
            printf("auth %d, count %d\n", (t_top[i] + 1), t_cnt[i]);
    }
}

/**
 * @brief Detertime the top number of authors
 * 
 * @param authors       list of authors in order of id (index)
 * @param authPerProc   size of authors
 * @param topAuth       list of top authors
 * @param topCounts     counts associated with author at index
 * @param top           size of top list
 */
void determineTop(int *authors, int *topAuth, int *topCounts)
{
    int currCount;
    for (int i = 0; i < VERT; i++)
    {
        currCount = authors[i];

        for (int j = 0; j < TOP; j++)
        {
            if (currCount > topCounts[j])
            {
                // replace smallest author at the end.
                topAuth[TOP - 1] = i;
                topCounts[TOP - 1] = currCount;

                sort(topAuth, topCounts, TOP);
                break;
            }
        }
    }
}

/**
 * @brief Bubble sort
 * 
 * @param id    ids assoc with vals
 * @param val   values to sort
 * @param len   list length
 */
void sort(int *id, int *val, int len)
{
    bool swapped;

    for (int i = 0; i < len - 1; i++)
    {
        swapped = false;
        for (int j = 0; j < len - i - 1; j++)
        {
            if (val[j] < val[j + 1])
            {
                swap(val[j], val[j + 1]);
                swap(id[j], id[j + 1]);
                swapped = true;
            }
        }

        if (!swapped)
            break;
    }
}

/**
 * @brief Get a time point
 * 
 * @return double   The time point generated
 */
double CLOCK()
{
    struct timespec t;
    clock_gettime(CLOCK_MONOTONIC, &t);
    return (t.tv_sec * 1000) + (t.tv_nsec * 1e-6);
}